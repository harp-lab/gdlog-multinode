#include <cstdint>
#include <cuda/std/chrono>
#include <iostream>
#include <vector>
// thrust use TBB
// #define THRUST_HOST_SYSTEM THRUST_HOST_SYSTEM_TBB

#include "../include/hashtrie.cuh"

#include "../include/exception.cuh"
#include "../include/lie.cuh"
#include "../include/print.cuh"
#include "../include/timer.cuh"

#include <execinfo.h>
#include <signal.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

void handler(int sig) {
    void *array[10];

    // get void*'s for all entries on the stack
    size_t size = backtrace(array, 10);
    char **strs = backtrace_symbols(array, size);
    for (int i = 0; i < size; i++) {
        printf("%s\n", strs[i]);
    }

    // print out all the frames to stderr
    fprintf(stderr, "Error: signal %d:\n", sig);
    backtrace_symbols_fd(array, size, STDERR_FILENO);
    exit(1);
}

const uint32_t TEST_CASE_SIZE = 10000 * 1000;
const uint32_t REPEAT = 100;
const uint32_t TEST_ARIRTY = 2;

void generate_random_data(std::vector<uint32_t> &data_column1) {
    for (int i = 0; i < data_column1.size(); i++) {
        data_column1[i] = rand() % TEST_CASE_SIZE;
    }
}

void test_load(std::vector<std::vector<uint32_t>> &data_columns,
               hisa::hisa_cpu &hashtrie_cpu) {
    hashtrie_cpu.load_vectical(data_columns);
    hashtrie_cpu.deduplicate();
    hashtrie_cpu.build_index();
}

void test_load_hisa(std::vector<uint32_t> &data_columns, Relation *rel) {
    int device_id;
    int number_of_sm;
    hipGetDevice(&device_id);
    hipDeviceGetAttribute(&number_of_sm, hipDeviceAttributeMultiprocessorCount,
                           device_id);
    int block_size, grid_size;
    block_size = 512;
    grid_size = 32 * number_of_sm;
    uint32_t rows = data_columns.size() / TEST_ARIRTY;
    load_relation(rel, "rel", TEST_ARIRTY, data_columns.data(), rows,
                  TEST_ARIRTY - 1, 0, grid_size, block_size);
}

void raw_vertical_to_horizontal(
    std::vector<std::vector<uint32_t>> &data_columns,
    std::vector<uint32_t> &data_columns_horizontal) {
    auto total_size = data_columns.size() * data_columns[0].size();
    for (int i = 0; i < data_columns[0].size(); i++) {
        for (int j = 0; j < data_columns.size(); j++) {
            data_columns_horizontal.push_back(data_columns[j][i]);
        }
    }
}

void testcase_deduplicate() {
    std::vector<std::vector<hisa::internal_data_type>> test_raw;
    test_raw.push_back({{1, 6, 3, 9, 1, 2, 3, 8}});
    test_raw.push_back({{3, 2, 3, 9, 3, 7, 3, 1}});
    // 3 8 1 1 3 2
    // 3 7 1 2 3 3
    hisa::hisa_cpu h(2);
    h.load_vectical(test_raw);

    h.deduplicate();
    for (int i = 0; i < 2; i++) {
        for (int j = 0; j < h.columns[i].size(); j++) {
            std::cout << h.columns[i].raw_data[j] << " ";
        }
        std::cout << std::endl;
    }
    // h.build_index();

    // h.print_all();
}

int main() {
    signal(SIGSEGV, handler);
    // generate 3 columns of random data
    std::vector<std::vector<uint32_t>> data_columns_vertical(TEST_ARIRTY);
    std::vector<uint32_t> data_columns_horizontal;

    for (int i = 0; i < TEST_ARIRTY; i++) {
        data_columns_vertical[i].resize(TEST_CASE_SIZE);
        generate_random_data(data_columns_vertical[i]);
    }
    raw_vertical_to_horizontal(data_columns_vertical, data_columns_horizontal);

    std::cout << "generate_random_data done" << std::endl;
    hisa::hisa_cpu hashtrie_cpu(TEST_ARIRTY);
    hashtrie_cpu.load_vectical(data_columns_vertical);
    hashtrie_cpu.build_index();

    testcase_deduplicate();

    uint64_t total_hisa_load_time = 0;
    uint64_t total_hashtrie_cpu_load_time = 0;
    uint64_t total_hash_time = 0;

    for (int i = 0; i < REPEAT; i++) {
        std::cout << "repeat: " << i << std::endl;
        // test load
        hisa::hisa_cpu hashtrie_cpu2(TEST_ARIRTY);
        auto start = std::chrono::high_resolution_clock::now();
        test_load(data_columns_vertical, hashtrie_cpu2);
        auto end = std::chrono::high_resolution_clock::now();
        auto duration =
            std::chrono::duration_cast<std::chrono::microseconds>(end - start);
        total_hashtrie_cpu_load_time += duration.count();
        total_hash_time += hashtrie_cpu2.hash_time;

        // test load hisa
        Relation *rel = new Relation();
        auto start_hisa = std::chrono::high_resolution_clock::now();
        test_load_hisa(data_columns_horizontal, rel);
        auto end_hisa = std::chrono::high_resolution_clock::now();
        auto duration_hisa =
            std::chrono::duration_cast<std::chrono::microseconds>(end_hisa -
                                                                  start_hisa);
        total_hisa_load_time += duration_hisa.count();
        rel->drop();
        delete rel;
    }

    std::cout << "total_hashtrie_cpu_load_time: "
              << total_hashtrie_cpu_load_time << std::endl;
    std::cout << "total hash time : " << total_hash_time << std::endl;
    std::cout << "total_hisa_load_time: " << total_hisa_load_time << std::endl;

    return 0;
}
