#include "hip/hip_runtime.h"
#include <iostream>
#include <mpi.h>
#include <thrust/execution_policy.h>
#include <thrust/reduce.h>
#include <thrust/scan.h>
#include <thrust/unique.h>
#include <vector>
#include <rmm/exec_policy.hpp>

#include "../../include/exception.cuh"
#include "../../include/print.cuh"
#include "../../include/relational_algebra.cuh"
#include "../../include/timer.cuh"


__global__ void get_join_result_size(GHashRelContainer *inner_table,
                                     GHashRelContainer *outer_table,
                                     int join_column_counts,
                                     TupleGenerator tp_gen, TupleFilter tp_pred,
                                     tuple_size_t *join_result_size) {
    u64 index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= outer_table->tuple_counts)
        return;
    u64 stride = blockDim.x * gridDim.x;

    for (tuple_size_t i = index; i < outer_table->tuple_counts; i += stride) {
        tuple_type outer_tuple = outer_table->tuples[i];

        tuple_size_t current_size = 0;
        join_result_size[i] = 0;
        u64 hash_val = prefix_hash(outer_tuple, outer_table->index_column_size);
        // the index value "pointer" position in the index hash table
        tuple_size_t index_position = hash_val % inner_table->index_map_size;
        bool index_not_exists = false;
        while (true) {
            if (inner_table->index_map[index_position].key == hash_val &&
                tuple_eq(
                    outer_tuple,
                    inner_table
                        ->tuples[inner_table->index_map[index_position].value],
                    outer_table->index_column_size)) {
                break;
            } else if (inner_table->index_map[index_position].key ==
                       EMPTY_HASH_ENTRY) {
                index_not_exists = true;
                break;
            }
            index_position = (index_position + 1) % inner_table->index_map_size;
        }
        if (index_not_exists) {
            continue;
        }
        // pull all joined elements
        tuple_size_t position = inner_table->index_map[index_position].value;
        while (true) {
            tuple_type cur_inner_tuple = inner_table->tuples[position];
            bool cmp_res = tuple_eq(inner_table->tuples[position], outer_tuple,
                                    join_column_counts);
            if (cmp_res) {
                // hack to apply filter
                // TODO: this will cause max arity of a relation is 20
                if (tp_pred.arity > 0) {
                    column_type tmp[10] = {0};
                    tp_gen(cur_inner_tuple, outer_tuple, tmp);
                    if (tp_pred(tmp)) {
                        current_size++;
                    }
                } else {
                    current_size++;
                }
            } else {
                break;
            }
            position = position + 1;
            if (position > inner_table->tuple_counts - 1) {
                // end of data arrary
                break;
            }
        }
        join_result_size[i] = current_size;
    }
}

__global__ void
get_join_result(GHashRelContainer *inner_table, GHashRelContainer *outer_table,
                int join_column_counts, TupleGenerator tp_gen,
                TupleFilter tp_pred, int output_arity,
                column_type *output_raw_data, tuple_size_t *res_count_array,
                tuple_size_t *res_offset, JoinDirection direction) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= outer_table->tuple_counts)
        return;

    int stride = blockDim.x * gridDim.x;

    for (tuple_size_t i = index; i < outer_table->tuple_counts; i += stride) {
        if (res_count_array[i] == 0) {
            continue;
        }
        tuple_type outer_tuple = outer_table->tuples[i];

        int current_new_tuple_cnt = 0;
        u64 hash_val = prefix_hash(outer_tuple, outer_table->index_column_size);
        // the index value "pointer" position in the index hash table
        tuple_size_t index_position = hash_val % inner_table->index_map_size;
        bool index_not_exists = false;
        while (true) {
            if (inner_table->index_map[index_position].key == hash_val &&
                tuple_eq(
                    outer_tuple,
                    inner_table
                        ->tuples[inner_table->index_map[index_position].value],
                    outer_table->index_column_size)) {
                break;
            } else if (inner_table->index_map[index_position].key ==
                       EMPTY_HASH_ENTRY) {
                index_not_exists = true;
                break;
            }
            index_position = (index_position + 1) % inner_table->index_map_size;
        }
        if (index_not_exists) {
            continue;
        }

        // pull all joined elements
        tuple_size_t position = inner_table->index_map[index_position].value;
        while (true) {
            // TODO: always put join columns ahead? could be various benefits
            // but memory is issue to mantain multiple copies
            bool cmp_res = tuple_eq(inner_table->tuples[position], outer_tuple,
                                    join_column_counts);
            if (cmp_res) {
                // tuple prefix match, join here
                tuple_type inner_tuple = inner_table->tuples[position];
                tuple_type new_tuple =
                    output_raw_data +
                    (res_offset[i] + current_new_tuple_cnt) * output_arity;

                // for (int j = 0; j < output_arity; j++) {
                // TODO: this will cause max arity of a relation is 20
                if (tp_pred.arity > 0) {
                    column_type tmp[20];
                    tp_gen(inner_tuple, outer_tuple, tmp);
                    if (tp_pred(tmp)) {
                        tp_gen(inner_tuple, outer_tuple, new_tuple);
                        current_new_tuple_cnt++;
                    }
                } else {
                    tp_gen(inner_tuple, outer_tuple, new_tuple);
                    current_new_tuple_cnt++;
                }
                if (current_new_tuple_cnt > res_count_array[i]) {
                    break;
                }
            } else {
                // bucket end
                break;
            }
            position = position + 1;
            if (position > (inner_table->tuple_counts - 1)) {
                // end of data arrary
                break;
            }
        }
    }
}

void RelationalJoin::operator()() {

    bool output_is_tmp = output_rel->tmp_flag;
    GHashRelContainer *inner;
    if (inner_ver == DELTA) {
        inner = inner_rel->delta;
    } else {
        inner = inner_rel->full;
    }
    GHashRelContainer *outer;
    if (outer_ver == DELTA) {
        outer = outer_rel->delta;
    } else if (outer_ver == FULL) {
        outer = outer_rel->full;
    } else {
        // temp relation can be outer relation
        outer = outer_rel->newt;
    }
    int output_arity = output_rel->arity;
    // GHashRelContainer* output = output_rel->newt;

    if (outer->tuples == nullptr || outer->tuple_counts == 0) {
        outer->tuple_counts = 0;
        return;
    }
    if (inner->tuples == nullptr || inner->tuple_counts == 0) {
        outer->tuple_counts = 0;
        return;
    }

    KernelTimer timer;
    // checkCuda(hipDeviceSynchronize());
    GHashRelContainer *inner_device;
    checkCuda(hipMalloc((void **)&inner_device, sizeof(GHashRelContainer)));
    checkCuda(hipMemcpy(inner_device, inner, sizeof(GHashRelContainer),
                         hipMemcpyHostToDevice));
    GHashRelContainer *outer_device;
    checkCuda(hipMalloc((void **)&outer_device, sizeof(GHashRelContainer)));
    checkCuda(hipMemcpy(outer_device, outer, sizeof(GHashRelContainer),
                         hipMemcpyHostToDevice));

    tuple_size_t *result_counts_array;
    checkCuda(hipMalloc((void **)&result_counts_array,
                         outer->tuple_counts * sizeof(tuple_size_t)));
    checkCuda(hipMemset(result_counts_array, 0,
                         outer->tuple_counts * sizeof(tuple_size_t)));

    // checkCuda(hipDeviceSynchronize());
    timer.start_timer();
    checkCuda(hipDeviceSynchronize());
    get_join_result_size<<<grid_size, block_size>>>(
        inner_device, outer_device, outer->index_column_size, tuple_generator,
        tuple_pred, result_counts_array);
    checkCuda(hipGetLastError());
    checkCuda(hipDeviceSynchronize());
    timer.stop_timer();
    this->detail_time[0] += timer.get_spent_time();

    timer.start_timer();
    tuple_size_t total_result_rows = 0;
    for (tuple_size_t i = 0; i < outer->tuple_counts; i = i + MAX_REDUCE_SIZE) {
        tuple_size_t reduce_size = MAX_REDUCE_SIZE;
        if (i + MAX_REDUCE_SIZE > outer->tuple_counts) {
            reduce_size = outer->tuple_counts - i;
        }
        tuple_size_t reduce_v =
            thrust::reduce(rmm::exec_policy(), result_counts_array + i,
                           result_counts_array + i + reduce_size, 0);
        total_result_rows += reduce_v;
        // checkCuda(hipDeviceSynchronize());
    }

    tuple_size_t *result_counts_offset;
    checkCuda(hipMalloc((void **)&result_counts_offset,
                         outer->tuple_counts * sizeof(tuple_size_t)));
    checkCuda(hipMemcpy(result_counts_offset, result_counts_array,
                         outer->tuple_counts * sizeof(tuple_size_t),
                         hipMemcpyDeviceToDevice));
    thrust::exclusive_scan(rmm::exec_policy(), result_counts_offset,
                           result_counts_offset + outer->tuple_counts,
                           result_counts_offset);

    timer.stop_timer();
    detail_time[1] += timer.get_spent_time();

    timer.start_timer();
    column_type *join_res_raw_data;
    u64 join_res_raw_data_mem_size =
        total_result_rows * output_arity * sizeof(column_type);
    checkCuda(
        hipMalloc((void **)&join_res_raw_data, join_res_raw_data_mem_size));
    checkCuda(hipMemset(join_res_raw_data, 0, join_res_raw_data_mem_size));
    get_join_result<<<grid_size, block_size>>>(
        inner_device, outer_device, outer->index_column_size, tuple_generator,
        tuple_pred, output_arity, join_res_raw_data, result_counts_array,
        result_counts_offset, LEFT);
    checkCuda(hipGetLastError());
    checkCuda(hipDeviceSynchronize());
    timer.stop_timer();
    detail_time[2] += timer.get_spent_time();
    checkCuda(hipFree(result_counts_array));
    checkCuda(hipFree(result_counts_offset));

    float load_relation_container_time[5] = {0, 0, 0, 0, 0};
    // // reload newt
    // free_relation(output_newt);
    // newt don't need index
    if (output_rel->newt->tuples == nullptr ||
        output_rel->newt->tuple_counts == 0) {
        if (disable_load) {
            return;
        }
        if (!output_is_tmp) {
            load_relation_container(
                output_rel->newt, output_arity, join_res_raw_data,
                total_result_rows, output_rel->index_column_size,
                output_rel->dependent_column_size, 0.8, grid_size, block_size,
                load_relation_container_time, true, false, false);
        } else {
            // temporary relation doesn't need index nor sort
            load_relation_container(
                output_rel->newt, output_arity, join_res_raw_data,
                total_result_rows, output_rel->index_column_size,
                output_rel->dependent_column_size, 0.8, grid_size, block_size,
                load_relation_container_time, true, true, false);
            output_rel->newt->tmp_flag = true;
        }
        checkCuda(hipDeviceSynchronize());
        detail_time[3] += load_relation_container_time[0];
        detail_time[4] += load_relation_container_time[1];
        detail_time[5] += load_relation_container_time[2];
    } else {
        // TODO: handle the case out put relation is temp relation
        // data in current newt, merge
        if (!output_is_tmp) {
            GHashRelContainer *newt_tmp = new GHashRelContainer(
                output_rel->arity, output_rel->index_column_size,
                output_rel->dependent_column_size);
            GHashRelContainer *old_newt = output_rel->newt;
            load_relation_container(
                newt_tmp, output_arity, join_res_raw_data, total_result_rows,
                output_rel->index_column_size,
                output_rel->dependent_column_size, 0.8, grid_size, block_size,
                load_relation_container_time, true, false, false);
            detail_time[3] += load_relation_container_time[0];
            detail_time[4] += load_relation_container_time[1];
            detail_time[5] += load_relation_container_time[2];
            RelationalUnion ru(newt_tmp, output_rel->newt);
            ru();
            output_rel->newt->fit();
            newt_tmp->free();
            delete newt_tmp;
        } else {
            // output relation is tmp relation, directly merge without sort
            GHashRelContainer *old_newt = output_rel->newt;
            column_type *newt_tmp_raw;
            u64 newt_tmp_raw_mem_size =
                (old_newt->tuple_counts + total_result_rows) *
                output_rel->arity * sizeof(column_type);
            tuple_size_t new_newt_counts =
                old_newt->tuple_counts + total_result_rows;
            checkCuda(
                hipMalloc((void **)&newt_tmp_raw, newt_tmp_raw_mem_size));
            checkCuda(hipMemcpy(newt_tmp_raw, old_newt->data_raw,
                                 old_newt->tuple_counts * old_newt->arity *
                                     sizeof(column_type),
                                 hipMemcpyDeviceToDevice));
            checkCuda(hipMemcpy(
                &(newt_tmp_raw[old_newt->tuple_counts * old_newt->arity]),
                join_res_raw_data,
                total_result_rows * output_rel->arity * sizeof(column_type),
                hipMemcpyDeviceToDevice));
            old_newt->free();
            checkCuda(hipFree(join_res_raw_data));
            load_relation_container(
                output_rel->newt, output_arity, newt_tmp_raw, new_newt_counts,
                output_rel->index_column_size,
                output_rel->dependent_column_size, 0.8, grid_size, block_size,
                load_relation_container_time, true, true, false);
            checkCuda(hipDeviceSynchronize())
        }

        detail_time[3] += load_relation_container_time[0];
        detail_time[4] += load_relation_container_time[1];
        detail_time[5] += load_relation_container_time[2];
        // print_tuple_rows(output_rel->newt, "join merge newt");
        // delete newt_tmp;
    }

    // print_tuple_rows(output_rel->newt, "output_newtr");
    // checkCuda(hipDeviceSynchronize());
    // std::cout << output_rel->name << " join result size " <<
    // output_rel->newt->tuple_counts <<std::endl;

    checkCuda(hipFree(inner_device));
    checkCuda(hipFree(outer_device));
}
