#include "hip/hip_runtime.h"

#include "../../include/exception.cuh"
#include "../../include/relational_algebra.cuh"

#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/remove.h>
#include <thrust/set_operations.h>



__global__ void
get_join_inner(MEntity *inner_index_map, tuple_size_t inner_index_map_size,
               tuple_size_t inner_tuple_counts, tuple_type *inner_tuples,
               tuple_type *outer_tuples, tuple_size_t outer_tuple_counts,
               int join_column_counts, bool *join_result_bitmap) {
    u64 index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= outer_tuple_counts)
        return;
    u64 stride = blockDim.x * gridDim.x;

    for (tuple_size_t i = index; i < outer_tuple_counts; i += stride) {
        tuple_type outer_tuple = outer_tuples[i];

        u64 hash_val = prefix_hash(outer_tuple, join_column_counts);
        // the index value "pointer" position in the index hash table
        tuple_size_t index_position = hash_val % inner_index_map_size;
        bool index_not_exists = false;
        while (true) {
            if (inner_index_map[index_position].key == hash_val &&
                tuple_eq(outer_tuple,
                         inner_tuples[inner_index_map[index_position].value],
                         join_column_counts)) {
                break;
            } else if (inner_index_map[index_position].key ==
                       EMPTY_HASH_ENTRY) {
                index_not_exists = true;
                break;
            }
            index_position = (index_position + 1) % inner_index_map_size;
        }
        if (index_not_exists) {
            continue;
        }
        // pull all joined elements
        tuple_size_t position = inner_index_map[index_position].value;
        while (true) {
            bool cmp_res = tuple_eq(inner_tuples[position], outer_tuple,
                                    join_column_counts);
            if (cmp_res) {
                join_result_bitmap[position] = true;
            } else {
                break;
            }
            position = position + 1;
            if (position > inner_tuple_counts - 1) {
                // end of data arrary
                break;
            }
        }
    }
}

void RelationalNegation::operator()() {
    GHashRelContainer *src;
    if (src_ver == DELTA) {
        src = src_rel->delta;
    } else if (src_ver == FULL) {
        src = src_rel->full;
    } else {
        src = src_rel->newt;
    }
    GHashRelContainer *negate;
    if (neg_ver == DELTA) {
        negate = neg_rel->delta;
    } else if (neg_ver == FULL) {
        negate = neg_rel->full;
    } else {
        negate = neg_rel->newt;
    }

    int jcc = neg_rel->index_column_size;

    thrust::device_vector<bool> result_bitmap_vec(src->tuple_counts, false);
    get_join_inner<<<grid_size, block_size>>>(
        src->index_map, src->index_map_size, src->tuple_counts, src->tuples,
        negate->tuples, negate->tuple_counts, jcc,
        result_bitmap_vec.data().get());
    
    std::cout << "Negation result bitmap: ";
    for (int i = 0; i < src->tuple_counts; i++) {
        std::cout << result_bitmap_vec[i] << " ";
    }

    auto new_tuple_end = thrust::remove_if(
        thrust::device, src->tuples, src->tuples + src->tuple_counts,
        result_bitmap_vec.begin(), thrust::identity<bool>());

    src->tuple_counts = new_tuple_end - src->tuples;
}
